#include "hip/hip_runtime.h"
#include "offline_channelizer.cuh"

void __global__ create_polyphase_input(hipfftComplex *inp, hipfftComplex *outp, int nchannel, int nslice)
{
    int slice_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (slice_id < nslice)
    {
        for (int channel_id = 0; channel_id < nchannel; channel_id++)
        {
            outp[channel_id * nslice + slice_id] = inp[(1 + slice_id) * nchannel - 1 - channel_id];
        }
    }
}

void __global__ multiply(hipfftComplex *inp, hipfftComplex *coeff, hipfftComplex *outp, int nsamples)
{
    int sample_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (sample_id < nsamples)
        outp[sample_id] = make_hipComplex(inp[sample_id].x * coeff[sample_id].x - inp[sample_id].y * coeff[sample_id].y, inp[sample_id].x * coeff[sample_id].y + inp[sample_id].y * coeff[sample_id].x);
}

channelizer::channelizer(int nchann, int nsl, complex<float>* coeff_arr)
{
    nchannel = nchann;
    nslice   = nsl;

    // Allocate GPU memory for filter coefficients.
    hipMalloc((void**) &coeff_fft_polyphaseform, sizeof(hipfftComplex)*nchannel*nslice);

    // Allocate GPU memory for internal buffer.
    hipMalloc((void**) &internal_buffer, sizeof(hipfftComplex)*nchannel*nslice);

    /*
     * Plan 1 : Take FFT along each row. There are nslice elements in each row.
     * There are nchannel rows.
     */
    istride_1 = 1;
    idist_1 = nslice;
    batch_1 = nchannel;
    ostride_1 = 1;
    odist_1 = nslice;
    n_1 = new int [1];
    *n_1 = nslice;
    inembed_1 = n_1;
    onembed_1 = n_1;

    /*
     * Plan 2 : Take IFFT along each row. There are nslice elements in each row.
     * There are nchannel rows.
     */
    // istride_2 = 1;
    // idist_2 = nslice;
    // batch_2 = nchannel;
    // ostride_2 = 1;
    // odist_2 = nslice;

    /*
     * Plan 3 : Take IFFT along each column. There are nslice elements in each row.
     * There are nchannel rows.
     */
    istride_2 = nslice;
    idist_2 = 1;
    batch_2 = nslice;
    ostride_2 = nslice;
    odist_2 = 1;
    n_2 = new int [1];
    *n_2 = nchannel;
    inembed_2 = n_2;
    onembed_2 = n_2;

    hipfftPlanMany(&plan_1, rank, n_1, inembed_1, istride_1, idist_1, onembed_1, ostride_1, odist_1, HIPFFT_C2C, batch_1);
    hipfftPlanMany(&plan_2, rank, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
}