#include "hip/hip_runtime.h"
// #include "../include/cinterface.cuh"
#include "../include/offline_channelizer.cuh"
#include "../include/offline_chann_C_interface.cuh"
#include <stdio.h>
#include <cmath>
#include <complex>
#include <chrono>
#include <iostream>
using namespace std::complex_literals;
using std::chrono::high_resolution_clock;
using std::chrono::steady_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::cyl_bessel_if;
using std::cout;
using std::endl;
using std::milli;
using std::complex;

float sinc(float x)
{
    return (x == 0.0) ? 1.0 : float(sin(x)/x);
}

int main()
{
    int Nsamples = 100000000;
    int Nch   = 1024;
    int Nslice = 2*1024*128;
    int Nproto = 100;
    float kbeta=9.6;
    vector<complex<float>> filter_function;
    for (int j=0; j<Nch*Nproto; j++)
    {
        float arg = Nproto / 2 + static_cast<float>(j + 1) / Nch;
        float darg = static_cast<float>(2 * j) / static_cast<float>(Nch*Nproto) - 1.0;
        float carg = kbeta * sqrt(1-darg*darg);
        try{
        float earg = cyl_bessel_if(0.0, carg) / cyl_bessel_if(0.0, kbeta);
        filter_function.push_back(complex<float>(earg, 0.0));
        }
        catch(int num)
        {
            cout << "Exception occured " << j << endl;
        }
    }
    chann* p_chann = chann_create(&filter_function[0], Nproto, Nch, Nslice);
    float* input = new float [Nch*(Nslice)];
    hipfftComplex* inp_c = new hipfftComplex [Nch * Nslice / 2];
    hipfftComplex* output_gpu;
    hipfftComplex* output_cpu;
    output_cpu = new hipfftComplex [Nch*Nslice];
    hipMalloc((void **)&output_gpu, sizeof(hipfftComplex) * Nch * Nslice);
    for (int k=0; k<2*Nsamples; k++)
    {
        float inp_arg = static_cast<float>(k / 2);
        if (k%2 == 0)
        {
            input[k] = sin(inp_arg);
        }
        else 
        {
            input[k] = sinc(2.0*inp_arg);
        }
    }
    cout << "---------------------------------------" << endl;
    float total_duration = 0.0;
    float total_duration_cpy = 0.0;
    int ntimes = 100;
    float milliseconds;
    for (int i=0; i<10;i++)
    {
        chann_process(p_chann, input, output_gpu, i);
        total_duration += reinterpret_cast<channelizer*>(p_chann)->time;
        total_duration_cpy += reinterpret_cast<channelizer*>(p_chann)->time_cpy;
    }
    transfer(output_gpu, output_cpu, 10);
    //Check that results are legitimate.
    for (int i=0; i< 10; i++)
    {
        cout << output_cpu[i].x << " " << output_cpu[i].y << endl;
    }
    std::cout << "Time taken in milliseconds to process " << Nsamples <<" samples into 1024 channels is " << (total_duration / 10) << " and time taken to copy is " << (total_duration_cpy / 10) << std::endl;
    chann_destroy(p_chann);
    delete [] input;
    delete [] output_cpu;
    hipFree(output_gpu);
}