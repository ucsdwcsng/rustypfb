#include "hip/hip_runtime.h"
// #include "../include/cinterface.cuh"
#include "../include/offline_channelizer.cuh"
#include <stdio.h>
#include <cmath>
#include <complex>
#include <chrono>
#include <iostream>
using namespace std::complex_literals;
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::cyl_bessel_if;
using std::cout;
using std::endl;

using std::complex;

float sinc(float x)
{
    return (x == 0.0) ? 1.0 : float(sin(x)/x);
}

int main()
{
    int Nsamples = 100000000;
    int Nch   = 1024;
    int Nslice = 1024*128;
    int Nproto = 100;
    float kbeta=9.6;
    vector<complex<float>> filter_function;
    for (int j=0; j<Nch*Nproto; j++)
    {
        float arg = Nproto / 2 + static_cast<float>(j + 1) / Nch;
        float darg = static_cast<float>(2 * j) / static_cast<float>(Nch*Nproto) - 1.0;
        float carg = kbeta * sqrt(1-darg*darg);
        try{
        float earg = cyl_bessel_if(0.0, carg) / cyl_bessel_if(0.0, kbeta);
        filter_function.push_back(complex<float>(earg, 0.0));
        }
        catch(int num)
        {
            cout << "Exception occured " << j << endl;
        }
        // cout << arg << " " << sinc(arg) << " " << darg << " " << carg << " " << earg <<endl;
        // float barg = sinc(arg) * cyl_bessel_if(0.0, ) / cyl_bessel_if(0.0, kbeta);cd
        
    }

    for (int k=0; k< 1280; k++)
    {
        cout << filter_function[k].real() << " " << filter_function[k].imag() << endl;
    }    
    auto obj_chann = channelizer(&filter_function[0]);

    complex<float>* input = new complex<float>[Nch*Nslice];
    complex<float>* output_gpu;
    hipMalloc((void **)&output_gpu, sizeof(hipfftComplex) * Nch * Nslice*2);
    for (int k=0; k<Nsamples; k++)
    {
        complex<float> t(sin(k), sinc(2.0 *k));
        input[k] = t;
    }
    complex<float>* output_cpu = new complex<float>[10];

    double total_duration = 0.0;
    int ntimes = 100;

    for (int i=0; i<ntimes; i++)
    {
        auto start = high_resolution_clock::now();
        obj_chann.process(input, output_gpu);
        auto end = high_resolution_clock::now();
        double f = duration<double>(end-start).count();
        hipMemcpy(output_cpu, output_gpu, sizeof(complex<float>)*10, hipMemcpyDeviceToHost);
        for (int i=0; i<10; i++)
    {
        cout << output_cpu[i].real() << " " << output_cpu[i].imag() << endl;  
    }
        cout << "-----------------------------------------" << endl;
        total_duration += f;
    }
    std::cout << "Time taken in seconds to process " << Nsamples <<" samples into 1024 channels is " << (total_duration / ntimes) << std::endl;
    delete [] input;
    hipFree(output_gpu);
    delete [] output_cpu;
}