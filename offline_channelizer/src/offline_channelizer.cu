#include "hip/hip_runtime.h"
#include "../include/offline_channelizer.cuh"
#include <iostream>
#include <omp.h>
#include <stdio.h>
using std::cout;
using std::endl;
using std::make_unique;


// const int NCHANNEL = 1024;
// const int NCHANNELHALF = 512;
// const int NSLICE   = 2*131072;
// const int NPROTO   = 100;
const int BLOCKCHANNELS = 32;
const int BLOCKSLICES = 32;
// const int GRIDCHANNELS = 32;
// const int GRIDSUBCHANNELS = 16;
// const int GRIDSLICES = 8192;
// const int HALFSUBCHANNELS = 16;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*
 * Create the FFT version of the coefficient filters for non-maximal decimation.
 */
void make_coeff_matrix(hipfftComplex* gpu, complex<float>* inp, int nproto, int nchannel, int nslice) {
    int nchannelhalf = nchannel / 2;
    for (int id = 0; id < nchannel * nproto; id++)
    {
        int tap_id = id / nchannel;
        int chann_id = id % nchannel;
        if (chann_id < nchannelhalf)
        {
            hipMemcpy(gpu + 2 * tap_id  + chann_id * nslice, inp + id, sizeof(hipfftComplex), hipMemcpyHostToDevice);
        }
        else 
        {
            hipMemcpy(gpu + 2 * tap_id + 1 + chann_id * nslice, inp + id, sizeof(hipfftComplex), hipMemcpyHostToDevice);   
        }
        // auto err_0 = hipGetLastError();
    }
    int istride = 1;
    int ostride = 1;
    int idist = nslice;
    int odist = nslice;
    int batch = nchannel;
    int* n = new int [1];
    *n = nslice;
    int* inembed = n;
    int* onembed = n;
    hipfftHandle plan;
    hipfftPlanMany(&plan, 1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, gpu, gpu, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    delete [] n;

}

void __global__ multiply(hipfftComplex* inp, hipfftComplex* coeff, hipfftComplex* output, int nchannel, int nslice, int griddim)
{
    int half = blockIdx.y;
    int input_xcoord = blockIdx.x * blockDim.x + threadIdx.x;
    int raw_ycoord = blockIdx.y * blockDim.y + threadIdx.y;
    int output_id = (nchannel - raw_ycoord) * nslice + input_xcoord;
    int input_ycoord;
    int inp_id;
    int coeff_id;
    if (half < (griddim / 2))
    {
        input_ycoord = raw_ycoord;
        inp_id = input_ycoord*nslice + input_xcoord;
        coeff_id = inp_id;
    }
    else
    {
        input_ycoord = raw_ycoord - (griddim / 2)*blockDim.y; //(blockIdx.y - HALFSUBCHANNELS) * blockDim.y + threadIdx.y;
        inp_id = input_ycoord*nslice + input_xcoord;
        coeff_id = (nchannel*nslice / 2) + inp_id;
    }
    hipfftComplex lhs = inp[inp_id];
    hipfftComplex rhs = coeff[coeff_id];
    output[output_id] = make_hipComplex(lhs.x* rhs.x - lhs.y * rhs.y, lhs.x * rhs.y + lhs.y * rhs.x);
}

void __global__ scale(hipfftComplex* inp, bool row, int nchannel, int nslice)
{
    int inp_id = (blockIdx.y * blockDim.y + threadIdx.y)*nslice + blockIdx.x * blockDim.x + threadIdx.x;
    if (row)
    {
        inp[inp_id] = make_hipComplex(inp[inp_id].x / static_cast<float>(nslice), inp[inp_id].y / static_cast<float>(nslice));
    }
    else
    {
        inp[inp_id] = make_hipComplex(inp[inp_id].x / static_cast<float>(nchannel), inp[inp_id].y / static_cast<float>(nchannel));
    }
}

void __global__ alias(hipfftComplex* inp, int nslice)
{
    int x_coord = blockIdx.x * blockDim.x + threadIdx.x;
    int y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    int id = y_coord * nslice + x_coord;
    int signx = (1 - 2*(x_coord %2));
    bool signy = (y_coord % 2 == 0);
    if (signy)
    {
        if (signx != 1)
        {
            inp[id] = make_hipComplex(-inp[id].x, -inp[id].y);
        }
    }
}

void __global__ club(float* inp, hipfftComplex* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int out_id = static_cast<int>(id / 2);
    // printf("%d\n", id);
    if (id < size)
    {
        // printf("Inside Kernel function\n");
        if (id%2 == 0)
        {
            output[out_id].x = inp[id];
        }
        // printf("%f %f\n", output[out_id].x, output[out_id].y);
        else 
        {
            output[out_id].y = inp[id];
        }
    }
}

channelizer::channelizer(complex<float> *coeff_arr, int npr, int nchan, int nsl)
: nchannel{nchan}, nslice{nsl}, nproto{npr}, gridchannels{nchan / BLOCKCHANNELS}, gridslices{nsl / BLOCKSLICES}
{
    // Allocate GPU memory for filter coefficients.
    hipMalloc((void **)&coeff_fft_polyphaseform, sizeof(hipfftComplex) * nchannel * nslice);

    // Allocate Pagelocked memory for input buffer on host
    hipHostMalloc((void **)&locked_buffer, sizeof(hipfftComplex) * (nchannel / 2) * nslice);

    // Allocate Pagelocked memory for interleaved input buffer on host
    hipHostMalloc((void **)&locked_buffer_interleaved, sizeof(float) * nchannel * nslice);

    // Allocate GPU memory for output buffer.
    hipMalloc((void **)&output_buffer, sizeof(hipfftComplex) * nchannel * nslice);

    // Allocate GPU memory for scratch buffer.
    hipMalloc((void **)&scratch_buffer, sizeof(hipfftComplex) * (nchannel / 2) * nslice);
    /*
     * Initial FFT of input
     */
    istride_0 = (nchannel / 2);
    ostride_0 = 1;
    idist_0 = 1;
    odist_0 = nslice;
    batch_0 = (nchannel / 2);
    n_0 = new int [1];
    *n_0 = nslice;
    inembed_0 = n_0;
    onembed_0 = n_0;
    hipfftPlanMany(&plan_0, 1, n_0, inembed_0, istride_0, idist_0, onembed_0, ostride_0, odist_0, HIPFFT_C2C, batch_0);

    /*
     * Channel IFFT of samples which have had the filter FFT multiplied to them.
     */
    istride_1 = 1;
    ostride_1 = 1;
    idist_1 = nslice;
    odist_1 = nslice;
    batch_1 = nchannel;
    n_1 = new int [1];
    *n_1 = nslice;
    inembed_1 = n_1;
    onembed_1 = n_1;
    hipfftPlanMany(&plan_1, 1, n_1, inembed_1, istride_1, idist_1, onembed_1, ostride_1, odist_1, HIPFFT_C2C, batch_1);


    /*
     * Final Downconversion IFFT. This applies an IFFT along the channel dimension.
     */
    istride_2 = nslice;
    ostride_2 = nslice;
    idist_2 = 1;
    odist_2 = 1;
    batch_2 = nslice;
    n_2 = new int [1];
    *n_2 = nchannel;
    inembed_2 = n_2;
    onembed_2 = n_2;
    hipfftPlanMany(&plan_2, 1, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
    make_coeff_matrix(coeff_fft_polyphaseform, coeff_arr, nproto, nchannel, nslice);
}

void channelizer::process(float* input, complex<float>* output)
{
    dim3 dimBlockMultiply(BLOCKSLICES, BLOCKCHANNELS);
    dim3 dimGridMultiply(gridslices, gridchannels);
    memcpy(locked_buffer_interleaved, input, sizeof(float)*nchannel*nslice);
    club<<<nslice, nchannel>>>(locked_buffer_interleaved, locked_buffer, nslice*nchannel);
    // auto err_0 = hipGetLastError();
    // cout << hipGetErrorString(err_0) << endl;
    hipfftExecC2C(plan_0, locked_buffer, scratch_buffer, HIPFFT_FORWARD);
    multiply<<<dimGridMultiply, dimBlockMultiply>>>(scratch_buffer, coeff_fft_polyphaseform, output_buffer, nchannel, nslice, gridchannels);
    // auto err_1 = hipGetLastError();
    // cout << hipGetErrorString(err_1) << endl;
    hipfftExecC2C(plan_1, output_buffer, output_buffer, HIPFFT_BACKWARD);
    scale<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, true, nchannel, nslice);
    // auto err_2 = hipGetLastError();
    // cout << hipGetErrorString(err_2) << endl;
    hipfftExecC2C(plan_2, output_buffer, output_buffer, HIPFFT_BACKWARD);
    scale<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, false, nchannel, nslice);
    alias<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, nslice);
    // auto err_3 = hipGetLastError();
    // cout << hipGetErrorString(err_3) << endl;
    hipMemcpy(output, output_buffer, sizeof(complex<float>)*nslice*nchannel, hipMemcpyDeviceToDevice);
}

channelizer::~channelizer()
{
    hipfftDestroy(plan_0);
    hipfftDestroy(plan_1);
    hipfftDestroy(plan_2);
    delete [] n_0;
    delete [] n_1;
    delete [] n_2;
    hipFree(coeff_fft_polyphaseform);
    hipFree(scratch_buffer);
    hipHostFree(locked_buffer);
    hipHostFree(locked_buffer_interleaved);
    hipFree(output_buffer);
}

// unique_ptr<channelizer> create_chann(vector<complex<float>> coeff_arr)
// {
//     return make_unique<channelizer>(&coeff_arr[0]);
// }