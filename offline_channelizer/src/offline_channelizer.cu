#include "hip/hip_runtime.h"
#include "../include/offline_channelizer.cuh"
#include <iostream>
#include <omp.h>
#include <stdio.h>
using std::cout;
using std::endl;
using std::make_unique;


const int NCHANNEL = 1024;
const int NCHANNELHALF = 512;
const int NSLICE   = 2*131072;
const int NPROTO   = 100;
const int BLOCKCHANNELS = 32;
const int BLOCKSLICES = 32;
const int GRIDCHANNELS = 32;
const int GRIDSUBCHANNELS = 16;
const int GRIDSLICES = 8192;
const int HALFSUBCHANNELS = 16;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*
 * Create the FFT version of the coefficient filters for non-maximal decimation.
 */
void make_coeff_matrix(hipfftComplex* gpu, complex<float>* inp) {
    for (int id = 0; id < NCHANNEL * NPROTO; id++)
    {
        int tap_id = id / NCHANNEL;
        int chann_id = id % NCHANNEL;
        if (chann_id < NCHANNELHALF)
        {
            hipMemcpy(gpu + 2 * tap_id  + chann_id * NSLICE, inp + id, sizeof(hipfftComplex), hipMemcpyHostToDevice);
        }
        else 
        {
            hipMemcpy(gpu + 2 * tap_id + 1 + chann_id * NSLICE, inp + id, sizeof(hipfftComplex), hipMemcpyHostToDevice);   
        }
        // auto err_0 = hipGetLastError();
    }
    int istride = 1;
    int ostride = 1;
    int idist = NSLICE;
    int odist = NSLICE;
    int batch = NCHANNEL;
    int* n = new int [1];
    *n = NSLICE;
    int* inembed = n;
    int* onembed = n;
    hipfftHandle plan;
    hipfftPlanMany(&plan, 1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, gpu, gpu, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    delete [] n;

}

void __global__ multiply(hipfftComplex* inp, hipfftComplex* coeff, hipfftComplex* output)
{
    int half = blockIdx.y;
    int input_xcoord = blockIdx.x * blockDim.x + threadIdx.x;
    int raw_ycoord = blockIdx.y * blockDim.y + threadIdx.y;
    int output_id = (NCHANNEL - raw_ycoord) * NSLICE + input_xcoord;
    int input_ycoord;
    int inp_id;
    int coeff_id;
    if (half < HALFSUBCHANNELS)
    {
        input_ycoord = raw_ycoord;
        inp_id = input_ycoord*NSLICE + input_xcoord;
        coeff_id = inp_id;
    }
    else
    {
        input_ycoord = raw_ycoord - HALFSUBCHANNELS*blockDim.y; //(blockIdx.y - HALFSUBCHANNELS) * blockDim.y + threadIdx.y;
        inp_id = input_ycoord*NSLICE + input_xcoord;
        coeff_id = NCHANNELHALF*NSLICE + inp_id;
    }
    hipfftComplex lhs = inp[inp_id];
    hipfftComplex rhs = coeff[coeff_id];
    output[output_id] = make_hipComplex(lhs.x* rhs.x - lhs.y * rhs.y, lhs.x * rhs.y + lhs.y * rhs.x);
}

void __global__ scale(hipfftComplex* inp, bool row)
{
    int inp_id = (blockIdx.y * blockDim.y + threadIdx.y)*NSLICE + blockIdx.x * blockDim.x + threadIdx.x;
    if (row)
    {
        inp[inp_id] = make_hipComplex(inp[inp_id].x / static_cast<float>(NSLICE), inp[inp_id].y / static_cast<float>(NSLICE));
    }
    else
    {
        inp[inp_id] = make_hipComplex(inp[inp_id].x / static_cast<float>(NCHANNEL), inp[inp_id].y / static_cast<float>(NCHANNEL));
    }
}

void __global__ alias(hipfftComplex* inp)
{
    int x_coord = blockIdx.x * blockDim.x + threadIdx.x;
    int y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    int id = y_coord * NSLICE + x_coord;
    int signx = (1 - 2*(x_coord %2));
    bool signy = (y_coord % 2 == 0);
    if (signy)
    {
        if (signx != 1)
        {
            inp[id] = make_hipComplex(-inp[id].x, -inp[id].y);
        }
    }
}

void __global__ club(float* inp, hipfftComplex* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int out_id = static_cast<int>(id / 2);
    // printf("%d\n", id);
    if (id < size)
    {
        // printf("Inside Kernel function\n");
        if (id%2 == 0)
        {
            output[out_id].x = inp[id];
        }
        // printf("%f %f\n", output[out_id].x, output[out_id].y);
        else 
        {
            output[out_id].y = inp[id];
        }
    }
}

channelizer::channelizer(complex<float> *coeff_arr)
{
    // Allocate GPU memory for filter coefficients.
    hipMalloc((void **)&coeff_fft_polyphaseform, sizeof(hipfftComplex) * NCHANNEL * NSLICE);

    // Allocate Pagelocked memory for input buffer on host
    hipHostMalloc((void **)&locked_buffer, sizeof(hipfftComplex) * NCHANNELHALF * NSLICE);

    // Allocate Pagelocked memory for interleaved input buffer on host
    hipHostMalloc((void **)&locked_buffer_interleaved, sizeof(float) * NCHANNEL * NSLICE);

    // Allocate GPU memory for output buffer.
    hipMalloc((void **)&output_buffer, sizeof(hipfftComplex) * NCHANNEL * NSLICE);

    // Allocate GPU memory for scratch buffer.
    hipMalloc((void **)&scratch_buffer, sizeof(hipfftComplex) * NCHANNELHALF * NSLICE);
    /*
     * Initial FFT of input
     */
    istride_0 = NCHANNELHALF;
    ostride_0 = 1;
    idist_0 = 1;
    odist_0 = NSLICE;
    batch_0 = NCHANNELHALF;
    n_0 = new int [1];
    *n_0 = NSLICE;
    inembed_0 = n_0;
    onembed_0 = n_0;
    hipfftPlanMany(&plan_0, 1, n_0, inembed_0, istride_0, idist_0, onembed_0, ostride_0, odist_0, HIPFFT_C2C, batch_0);

    /*
     * Channel IFFT of samples which have had the filter FFT multiplied to them.
     */
    istride_1 = 1;
    ostride_1 = 1;
    idist_1 = NSLICE;
    odist_1 = NSLICE;
    batch_1 = NCHANNEL;
    n_1 = new int [1];
    *n_1 = NSLICE;
    inembed_1 = n_1;
    onembed_1 = n_1;
    hipfftPlanMany(&plan_1, 1, n_1, inembed_1, istride_1, idist_1, onembed_1, ostride_1, odist_1, HIPFFT_C2C, batch_1);


    /*
     * Final Downconversion IFFT. This applies an IFFT along the channel dimension.
     */
    istride_2 = NSLICE;
    ostride_2 = NSLICE;
    idist_2 = 1;
    odist_2 = 1;
    batch_2 = NSLICE;
    n_2 = new int [1];
    *n_2 = NCHANNEL;
    inembed_2 = n_2;
    onembed_2 = n_2;
    hipfftPlanMany(&plan_2, 1, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
    make_coeff_matrix(coeff_fft_polyphaseform, coeff_arr);
}

// void channelizer::process(complex<float>* input, complex<float>* output)
// {
//     dim3 dimBlockMultiply(BLOCKSLICES, BLOCKCHANNELS);
//     dim3 dimGridMultiply(GRIDSLICES, GRIDCHANNELS);
//     memcpy(locked_buffer, input, sizeof(hipfftComplex)*NCHANNELHALF*NSLICE);
//     hipfftExecC2C(plan_0, locked_buffer, scratch_buffer, HIPFFT_FORWARD);
//     multiply<<<dimGridMultiply, dimBlockMultiply>>>(scratch_buffer, coeff_fft_polyphaseform, output_buffer);
//     hipfftExecC2C(plan_1, output_buffer, output_buffer, HIPFFT_BACKWARD);
//     scale<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, true);
//     hipfftExecC2C(plan_2, output_buffer, output_buffer, HIPFFT_BACKWARD);
//     scale<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, false);
//     alias<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer);
//     hipMemcpy(output, output_buffer, sizeof(complex<float>)*NSLICE*NCHANNEL, hipMemcpyDeviceToDevice);
// }
void channelizer::process(float* input, complex<float>* output)
{
    dim3 dimBlockMultiply(BLOCKSLICES, BLOCKCHANNELS);
    dim3 dimGridMultiply(GRIDSLICES, GRIDCHANNELS);
    memcpy(locked_buffer_interleaved, input, sizeof(float)*NCHANNEL*NSLICE);
    club<<<NSLICE, NCHANNEL>>>(locked_buffer_interleaved, locked_buffer, NSLICE*NCHANNEL);
    // auto err_0 = hipGetLastError();
    // cout << hipGetErrorString(err_0) << endl;
    hipfftExecC2C(plan_0, locked_buffer, scratch_buffer, HIPFFT_FORWARD);
    multiply<<<dimGridMultiply, dimBlockMultiply>>>(scratch_buffer, coeff_fft_polyphaseform, output_buffer);
    // auto err_1 = hipGetLastError();
    // cout << hipGetErrorString(err_1) << endl;
    hipfftExecC2C(plan_1, output_buffer, output_buffer, HIPFFT_BACKWARD);
    scale<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, true);
    // auto err_2 = hipGetLastError();
    // cout << hipGetErrorString(err_2) << endl;
    hipfftExecC2C(plan_2, output_buffer, output_buffer, HIPFFT_BACKWARD);
    scale<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer, false);
    alias<<<dimGridMultiply, dimBlockMultiply>>>(output_buffer);
    // auto err_3 = hipGetLastError();
    // cout << hipGetErrorString(err_3) << endl;
    hipMemcpy(output, output_buffer, sizeof(complex<float>)*NSLICE*NCHANNEL, hipMemcpyDeviceToDevice);
}

channelizer::~channelizer()
{
    hipfftDestroy(plan_0);
    hipfftDestroy(plan_1);
    hipfftDestroy(plan_2);
    delete [] n_0;
    delete [] n_1;
    delete [] n_2;
    hipFree(coeff_fft_polyphaseform);
    hipFree(scratch_buffer);
    hipHostFree(locked_buffer);
    hipHostFree(locked_buffer_interleaved);
    hipFree(output_buffer);
}

unique_ptr<channelizer> create_chann(vector<complex<float>> coeff_arr)
{
    return make_unique<channelizer>(&coeff_arr[0]);
}