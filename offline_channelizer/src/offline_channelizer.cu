#include "hip/hip_runtime.h"
#include "../include/offline_channelizer.cuh"
#include "/opt/asmlib/asmlib.h"
#include <iostream>
#include <vector>
#include <omp.h>
#include <stdio.h>
// #include <string.h>
#include <chrono>
// #include <immintrin.h>
using std::cout;
using std::endl;
using std::make_unique;
using std::vector;
using std::chrono::steady_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::milli;


// const int NCHANNEL = 1024;
// const int NCHANNELHALF = 512;
// const int NSLICE   = 2*131072;
// const int NPROTO   = 100;
const int BLOCKCHANNELS = 32;
const int BLOCKSLICES = 32;
// const int GRIDCHANNELS = 32;
// const int GRIDSUBCHANNELS = 16;
// const int GRIDSLICES = 8192;
// const int HALFSUBCHANNELS = 16;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
 * Create the FFT version of the coefficient filters for non-maximal decimation.
 */
void make_coeff_matrix(hipfftComplex* gpu, complex<float>* inp, int nproto, int nchannel, int nslice) {
    int nchannelhalf = nchannel / 2;
    for (int id = 0; id < nchannel * nproto; id++)
    {
        int tap_id = id / nchannel;
        int chann_id = id % nchannel;
        if (chann_id < nchannelhalf)
        {
            hipMemcpy(gpu + 2 * tap_id  + chann_id * nslice, inp + id, sizeof(hipfftComplex), hipMemcpyHostToDevice);
        }
        else 
        {
            hipMemcpy(gpu + 2 * tap_id + 1 + chann_id * nslice, inp + id, sizeof(hipfftComplex), hipMemcpyHostToDevice);   
        }
        // auto err_0 = hipGetLastError();
    }
    int istride = 1;
    int ostride = 1;
    int idist = nslice;
    int odist = nslice;
    int batch = nchannel;
    int* n = new int [1];
    *n = nslice;
    int* inembed = n;
    int* onembed = n;
    hipfftHandle plan;
    hipfftPlanMany(&plan, 1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, gpu, gpu, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    delete [] n;

}

void __global__ multiply(hipfftComplex* inp, hipfftComplex* coeff, hipfftComplex* output, int nchannel, int nslice, int griddim)
{
    int half         = blockIdx.y;
    int input_xcoord = blockIdx.x * blockDim.x + threadIdx.x;
    int raw_ycoord   = blockIdx.y * blockDim.y + threadIdx.y;
    int output_id    = raw_ycoord * nslice + input_xcoord;
    // int coeff_id     = output_id;
    int input_ycoord;
    int input_id;
    if (half < (griddim / 2))
    {
        input_ycoord = raw_ycoord;
        // coeff_id     = input_ycoord*nslice + input_xcoord;
    }
    else
    {
        input_ycoord = raw_ycoord - (griddim / 2)*blockDim.y; //(blockIdx.y - HALFSUBCHANNELS) * blockDim.y + threadIdx.y;
        // coeff_id     = (nchannel*nslice / 2) + input_ycoord*nslice + input_xcoord;
    }
    input_id         = (nchannel / 2 - input_ycoord)*nslice + input_xcoord;
    hipfftComplex lhs = inp[input_id];
    hipfftComplex rhs = coeff[output_id];
    output[output_id] = make_hipComplex(lhs.x*rhs.x - lhs.y*rhs.y, lhs.x*rhs.y + lhs.y*rhs.x);
}

void __global__ scale(hipfftComplex* inp, bool row, int nchannel, int nslice)
{
    int inp_id = (blockIdx.y * blockDim.y + threadIdx.y)*nslice + blockIdx.x * blockDim.x + threadIdx.x;
    if (row)
    {
        inp[inp_id] = make_hipComplex(inp[inp_id].x / static_cast<float>(nslice), inp[inp_id].y / static_cast<float>(nslice));
    }
    else
    {
        inp[inp_id] = make_hipComplex(inp[inp_id].x / static_cast<float>(nchannel), inp[inp_id].y / static_cast<float>(nchannel));
    }
}

void __global__ alias(hipfftComplex* inp, int nslice)
{
    int x_coord = blockIdx.x * blockDim.x + threadIdx.x;
    int y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    int id = y_coord * nslice + x_coord;
    int signx = (1 - 2*(x_coord %2));
    bool signy = (y_coord % 2 == 0);
    if (signy)
    {
        if (signx != 1)
        {
            inp[id] = make_hipComplex(-inp[id].x, -inp[id].y);
        }
    }
}

// This reshape function has been tested to be correct
void __global__ reshape(hipfftComplex* inp, hipfftComplex* output, int nchannel, int nslice)
{
    __shared__ hipfftComplex tile[BLOCKCHANNELS][BLOCKSLICES];
    int input_x_coord = blockIdx.x * blockDim.x + threadIdx.x;
    int input_y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    auto inter = inp + (nchannel / 2)*input_y_coord + input_x_coord;
    tile[threadIdx.x][threadIdx.y] = *inter;
    __syncthreads();
    int output_grid_y_coord = (blockIdx.x * blockDim.x + threadIdx.y) * nslice;
    int output_grid_x_coord = blockIdx.y * blockDim.y + threadIdx.x;
    auto outer = output + output_grid_y_coord + output_grid_x_coord;
    (*outer) = tile[threadIdx.y][threadIdx.x];
}

void __global__ fft_shift(hipfftComplex* inp, int nslice, bool row)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy*nslice + idx;
    int sign = row ? (1-2*(idx % 2)) : (1-2*(idy % 2));
    inp[id] = make_hipComplex(sign*inp[id].x, sign*inp[id].y);
}

// // This is a very slow CUDA kernel, because of strided access
// void __global__ club(float* inp, hipfftComplex* output, int size)
// {
//     int id = blockIdx.x * blockDim.x + threadIdx.x;
//     int out_id = static_cast<int>(id / 2);
//     // printf("%d\n", id);
//     if (id < size)
//     {
//         // printf("Inside Kernel function\n");
//         if (id%2 == 0)
//         {
//             output[out_id].x = inp[id];
//         }
//         // printf("%f %f\n", output[out_id].x, output[out_id].y);
//         else 
//         {
//             output[out_id].y = inp[id];
//         }
//     }
// }

// void __global__ club_fromstream(float* real, float* imag, hipfftComplex* output, int size)
// {
//     int id = blockIdx.x * blockDim.x + threadIdx.x;
//     if (id < size)
//     {
//         output[id] = make_hipComplex(real[id], imag[id]);
//     }
// }

// void __global__ declub_fromstream(float* input, float* real, float* imag, int size, int* mask)
// {
//     int id = blockIdx.x * blockDim.x + threadIdx.x;
//     if (id < size)
//     {
//         if (id%2 == 0)
//         {
//             real[mask[id]] = input[id];
//         }
//         else
//         {
//             imag[mask[id]] = input[id];
//         }
//     }
// }

channelizer::channelizer(complex<float> *coeff_arr, int npr, int nchan, int nsl)
: nchannel{nchan}, nslice{nsl}, nproto{npr}, gridchannels{nchan / BLOCKCHANNELS}, gridslices{nsl / BLOCKSLICES} //, input_buffer(nchannel*nslice / 2)
{
    // Create CUDA events to calculate the running time on the default stream.
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // Allocate GPU memory for filter coefficients.
    hipMalloc((void **)&coeff_fft_polyphaseform, sizeof(hipfftComplex) * nchannel * nslice);

    // Allocate GPU memory to hold the input on the GPU.
    hipMalloc((void**)&input_buffer, sizeof(hipfftComplex)*nchannel*nslice/2);
    
    // Allocate GPU memory to hold reshaped input.
    hipMalloc((void **)&reshaped_buffer, sizeof(hipfftComplex) * (nchannel / 2) * nslice);

    // Allocate GPU memory to hold channelizer outputs.
    hipMalloc((void **)&output_buffer, sizeof(hipfftComplex) * nchannel * nslice);

    // Allocate GPU memory to hold intermediate scratch results.
    hipMalloc((void **)&scratch_buffer, sizeof(hipfftComplex) * (nchannel / 2) * nslice);

    /*
     * Initial FFT of input
     */
    istride_0 = 1;
    ostride_0 = 1;
    idist_0 = nslice;
    odist_0 = nslice;
    batch_0 = (nchannel / 2);
    n_0 = new int [1];
    *n_0 = nslice;
    inembed_0 = n_0;
    onembed_0 = n_0;
    hipfftPlanMany(&plan_0, 1, n_0, inembed_0, istride_0, idist_0, onembed_0, ostride_0, odist_0, HIPFFT_C2C, batch_0);

    /*
     * Channel IFFT of samples which have had the filter FFT multiplied to them.
     */
    istride_1 = 1;
    ostride_1 = 1;
    idist_1 = nslice;
    odist_1 = nslice;
    batch_1 = nchannel;
    n_1 = new int [1];
    *n_1 = nslice;
    inembed_1 = n_1;
    onembed_1 = n_1;
    hipfftPlanMany(&plan_1, 1, n_1, inembed_1, istride_1, idist_1, onembed_1, ostride_1, odist_1, HIPFFT_C2C, batch_1);


    /*
     * Final Downconversion IFFT. This applies an IFFT along the channel dimension.
     */
    istride_2 = nslice;
    ostride_2 = nslice;
    idist_2 = 1;
    odist_2 = 1;
    batch_2 = nslice;
    n_2 = new int [1];
    *n_2 = nchannel;
    inembed_2 = n_2;
    onembed_2 = n_2;
    hipfftPlanMany(&plan_2, 1, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
    make_coeff_matrix(coeff_fft_polyphaseform, coeff_arr, nproto, nchannel, nslice);
}

// input will be page locked on Host
void channelizer::process(float* input, hipfftComplex* output)
{
    dim3 dimBlock(BLOCKCHANNELS, BLOCKSLICES);
    dim3 dimGridMultiply(gridslices, gridchannels);
    dim3 dimGridReshape(gridchannels / 2, gridslices);
    // float duration_;
    // time = 0.0;
    // hipEventRecord(start,0);
    hipMemcpy(input_buffer, input, sizeof(hipfftComplex)*nchannel*nslice / 2, hipMemcpyHostToDevice);
    reshape<<<dimGridReshape, dimBlock>>>(input_buffer, reshaped_buffer, nchannel, nslice);
    hipfftExecC2C(plan_0, reshaped_buffer, scratch_buffer, HIPFFT_FORWARD);
    multiply<<<dimGridMultiply, dimBlock>>>(scratch_buffer, coeff_fft_polyphaseform, output_buffer, nchannel, nslice, gridchannels);
    hipfftExecC2C(plan_1, output_buffer, output_buffer, HIPFFT_BACKWARD);
    scale<<<dimGridMultiply, dimBlock>>>(output_buffer, true, nchannel, nslice);
    // fft_shift<<<dimGridMultiply, dimBlock>>>(output_buffer, nslice, false);
    hipfftExecC2C(plan_2, output_buffer, output_buffer, HIPFFT_BACKWARD);
    scale<<<dimGridMultiply, dimBlock>>>(output_buffer, false, nchannel, nslice);
    alias<<<dimGridMultiply, dimBlock>>>(output_buffer, nslice);
    hipMemcpy(output, output_buffer, sizeof(hipfftComplex)*nslice*nchannel, hipMemcpyDeviceToDevice);
    // hipEventRecord(stop,0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&duration_, start, stop);
    // time += duration_;
}

channelizer::~channelizer()
{
    hipfftDestroy(plan_0);
    hipfftDestroy(plan_1);
    hipfftDestroy(plan_2);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete [] n_0;
    delete [] n_1;
    delete [] n_2;
    hipFree(input_buffer);
    hipFree(coeff_fft_polyphaseform);
    hipFree(scratch_buffer);
    hipFree(reshaped_buffer);
    hipFree(output_buffer);
}