#include "hip/hip_runtime.h"
#include "../include/revert.cuh"
#include <vector>
#include <cmath>
using std::vector;

box::box(int a, int b, int c, int d, int e)
    : start_time{a}, stop_time{b}, start_chann{c}, stop_chann{d}, box_id{e}
{
}

box::box()
    : box(0, 0, 0, 0, 0) {}

synthesizer::synthesizer(int chann, int tap, int slice)
    : nchannel{chann}, ntaps{tap}, nslice{slice}
{
    input_plans = new hipfftHandle[36];
    downconvert_plans = new hipfftHandle[36];
    int istride = nslice;
    int ostride = nslice;
    int idist = 1;
    int odist = 1;
    vector<int> channel_vec{chann / 8, chann / 4, chann / 2};
    vector<int> slice_vec{slice / 8, slice / 4, slice / 2};
    for (int chann_dim = 0; chann_dim < 6; chann_dim++)
    {
        for (int slice_dim = 0; slice_dim < 6; slice_dim++)
        {
            hipfftPlanMany(&input_plans[6 * chann_dim + slice_dim], 1, &slice_vec[slice_dim], &slice_vec[slice_dim], 1, 
            slice_vec[slice_dim], &slice_vec[slice_dim], 1, slice_vec[slice_dim], HIPFFT_C2C, channel_vec[chann_dim]);
        }
    }
    for (int chann_dim = 0; chann_dim < 6; chann_dim++)
    {
        for (int slice_dim = 0; slice_dim < 6; slice_dim++)
        {
            hipfftPlanMany(&downconvert_plans[6 * chann_dim + slice_dim], 1, &channel_vec[chann_dim], &channel_vec[chann_dim], slice_vec[slice_dim], 
            1, &channel_vec[chann_dim], channel_vec[chann_dim], 1, HIPFFT_C2C, slice_vec[slice_dim]);
        }
    }

}

synthesizer::~synthesizer()
{
    for (int ind = 0; ind < 36; ind++)
    {
        hipfftDestroy(input_plans[ind]);
        hipfftDestroy(downconvert_plans[ind]);
    }
    delete[] input_plans;
    delete[] downconvert_plans;
}

float __device__ filter_value(int index, int nchannel, int taps)
{
    return cyl_bessel_i0f(static_cast<float>(index));
}

void synthesizer::revert(hipfftComplex *input, box* Box, hipfftComplex *scratch, hipfftComplex *output, int taps, int nboxes)
{
    // auto start_channel = curr_box.start_chann;
    // auto end_channel = curr_box.stop_chann;
    // auto start_time = curr_box.start_time;
    // auto end_time = curr_box.stop_time;

    // auto area = (end_time - start_time)*(end_channel - start_channel);

    // int padded_channel = (int)(log2(((32 * (end_channel - start_channel)) / nchannel) + 1));
    // int padded_slice = (int)(log2(((32 * (end_time - start_time)) / nslice) + 1));

    // auto full_channel = (int)pow(2, padded_channel);
    // int scratch_start_chann = (full_channel-(end_channel - start_channel)) / 2;

    // hipMemcpy2D(scratch + nslice * scratch_start_chann, nslice, input + start_channel * nslice, nslice, (end_time - start_time) * sizeof(hipfftComplex), end_channel - start_channel, hipMemcpyDeviceToDevice);
    // hipfftExecC2C(large_plans[6*padded_channel + padded_slice], scratch, scratch, HIPFFT_FORWARD);
    // synthesize<<<end_time - start_time, area, full_channel>>>(scratch, Box+boxind, output, taps);
}


void __global__ synthesize(hipfftComplex *input, box *Box, hipfftComplex *output, int taps)
{
    int inp_chann_id = blockDim.z * blockIdx.z + threadIdx.z;
    int inp_slice_id = blockDim.x * blockIdx.x + threadIdx.x;
    int outp_slice_id = blockDim.y * blockIdx.y + threadIdx.y;

    int nchannel = Box->stop_chann - Box->start_chann;
    int nslice = Box->stop_time - Box->start_time;

    if (inp_slice_id <= outp_slice_id)
    {
        atomicAdd(&output[outp_slice_id].x, filter_value(outp_slice_id - inp_slice_id, nchannel, taps) * input[inp_slice_id].x);
        atomicAdd(&output[outp_slice_id].y, filter_value(outp_slice_id - inp_slice_id, nchannel, taps) * input[inp_slice_id].y);
    }
}