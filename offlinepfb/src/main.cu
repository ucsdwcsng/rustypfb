#include "hip/hip_runtime.h"
#include "../include/offline_channelizer.cuh"
#include "../include/offline_chann_C_interface.cuh"
#include "../include/revert.cuh"
// #include "/opt/asmlib/asmlib.h"
// #include <string.h>
#include <stdio.h>
#include <cmath>
#include <complex>
#include <chrono>
#include <iostream>

using namespace std::complex_literals;
using std::chrono::high_resolution_clock;
using std::chrono::steady_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::cyl_bessel_if;
using std::cout;
using std::endl;
using std::milli;
using std::complex;

float sinc(float x)
{
    return (x == 0.0) ? 1.0 : float(sin(x)/x);
}

void time_test(chann* p_chann, float* input, hipfftComplex* output, int ntimes, float &time)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;
    hipEventRecord(start);
    for (int i=0; i < ntimes; i++)
    {
        chann_process(p_chann, input, output);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    time += duration;
}

int main()
{
    // int Nsamples = 100000000;
    // const int Nch   = 1024;
    // const int Nslice = 2*1024*128;
    // int Nproto = 100;
    // float kbeta=9.6;
    // vector<complex<float>> filter_function;
    // for (int j=0; j<Nch*Nproto; j++)
    // {
    //     float arg = Nproto / 2 + static_cast<float>(j + 1) / Nch;
    //     float darg = static_cast<float>(2 * j) / static_cast<float>(Nch*Nproto) - 1.0;
    //     float carg = kbeta * sqrt(1-darg*darg);
    //     try{
    //     float earg = cyl_bessel_if(0.0, carg) / cyl_bessel_if(0.0, kbeta);
    //     filter_function.push_back(complex<float>(earg, 0.0));
    //     }
    //     catch(int num)
    //     {
    //         cout << "Exception occured " << j << endl;
    //     }
    // }
    // chann* p_chann = chann_create(&filter_function[0], Nproto, Nch, Nslice);
    // float* input = new float [Nch*(Nslice)];
    // // hipfftComplex* inp_c = new hipfftComplex [Nch * Nslice / 2];
    // hipfftComplex* output_gpu;
    // hipfftComplex* output_cpu;
    // output_cpu = new hipfftComplex [Nch*Nslice];
    // hipMalloc((void **)&output_gpu, sizeof(hipfftComplex) * Nch * Nslice);
    // hipHostRegister(input, sizeof(float)*Nch*Nslice, hipHostRegisterMapped);
    // for (int k=0; k<2*Nsamples; k++)
    // {
    //     float inp_arg = static_cast<float>(k / 2);
    //     if (k%2 == 0)
    //     {
    //         input[k] = sin(inp_arg);
    //     }
    //     else 
    //     {
    //         input[k] = sinc(2.0*inp_arg);
    //     }
    // }
    // cout << "---------------------------------------" << endl;
    // float time;
    // time_test(p_chann, input, output_gpu, 50, time);
    // cout << "Channelization of " << Nsamples << " into 1024 channels takes " << time / 50 << " in milliseconds" << endl; 
    // chann_destroy(p_chann);
    // delete [] input;
    // delete [] output_cpu;
    // hipFree(output_gpu);

    // int Nproto = 128;
    // float kbeta = 10.2;
    // int Nch = 1024;
    // auto now = steady_clock::now();
    // for (int i=0; i<100; i++)
    // {
    //     float arg = Nproto / 2 + static_cast<float>(i + 1) / Nch;
    //     float darg = static_cast<float>(2 * i) / static_cast<float>(Nch*Nproto) - 1.0;
    //     float carg = kbeta * sqrt(1-darg*darg);
    //     float earg = cyl_bessel_if(0.0, carg) / cyl_bessel_if(0.0, kbeta);
    // }
    // auto end = steady_clock::now();
    // auto elapsed_time = duration<float, milli>(end - now).count();

    // cout << elapsed_time / 100 << endl;

    // auto box_collection = new box [16];

    // box* box_gpu;
    // hipMalloc((void**)&box_gpu, sizeof(box)*16);

    // hipMemcpy(box_gpu, box_collection, sizeof(box)*16, hipMemcpyHostToDevice);

    // test<<<4, 4>>>(box_gpu);
    // hipMemcpy(box_collection, box_gpu, sizeof(box)*16, hipMemcpyDeviceToHost);

    // for (int i=0; i< 16; i++)
    // {
    //     cout << box_collection[i].start_time << " " << box_collection[i].start_chann << " " << box_collection[i].stop_time << " " << box_collection[i].stop_chann << endl;
    // }

    hipfftHandle* plans;
    plans = new hipfftHandle [100];
    auto istrides = new int [100];
    auto ostrides = new int [100];
    auto idists   = new int [100];
    auto odists   = new int [100];
    auto batches  = new int [100];
    auto n = new int [100];
    
    for (int i=0; i < 100; i++)
    {
        istrides[i] = 2*i;
        ostrides[i] = 2*i;
        idists[i] = 4*i;
        odists[i] = 4*i;
        batches[i] = 1;
        n[i] = 2*(i%10+1);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float my_duration;
    hipEventRecord(start);
    for (int i=0; i<100; i++)
    {
        hipfftPlanMany(&plans[i], 1, n + i, n+i, istrides[i], idists[i], n+i, ostrides[i], odists[i], HIPFFT_C2C, batches[i]);
    }

    for (int i=0; i<100; i++)
    {
        hipfftDestroy(plans[i]);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&my_duration, start, stop);

    cout << "Initializing 100 plans and destroying them takes " << my_duration / 100 << " in milliseconds" << endl;
    
    delete [] plans;
    delete [] istrides;
    delete [] ostrides;
    delete [] idists;
    delete [] odists;
    delete [] batches;
    delete [] n;

}