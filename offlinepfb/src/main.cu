#include "hip/hip_runtime.h"
// #include "../include/revert.cuh"
#include "../include/offline_chann_C_interface.cuh"
// #include "/opt/asmlib/asmlib.h"
// #include <string.h>
#include <stdio.h>
#include <cmath>
#include <complex>
#include <chrono>
#include <iostream>

using namespace std::complex_literals;
using std::chrono::high_resolution_clock;
using std::chrono::steady_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::cyl_bessel_if;
using std::cout;
using std::endl;
using std::milli;
using std::complex;

float sinc(float x)
{
    return (x == 0.0) ? 1.0 : float(sin(x)/x);
}

void time_test(chann* p_chann, float* input, hipfftComplex* output, int ntimes, float &time)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float duration;
    hipEventRecord(start);
    for (int i=0; i < ntimes; i++)
    {
        chann_process(p_chann, input, output);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&duration, start, stop);
    time += duration;
}

int main()
{
    int Nsamples = 100000000;
    const int Nch   = 1024;
    const int Nslice = 2*1024*128;
    int Nproto = 100;
    float kbeta=9.6;
    vector<complex<float>> filter_function;
    for (int j=0; j<Nch*Nproto; j++)
    {
        float arg = Nproto / 2 + static_cast<float>(j + 1) / Nch;
        float darg = static_cast<float>(2 * j) / static_cast<float>(Nch*Nproto) - 1.0;
        float carg = kbeta * sqrt(1-darg*darg);
        try{
        float earg = cyl_bessel_if(0.0, carg) / cyl_bessel_if(0.0, kbeta);
        filter_function.push_back(complex<float>(earg, 0.0));
        }
        catch(int num)
        {
            cout << "Exception occured " << j << endl;
        }
    }
    chann* p_chann = chann_create(&filter_function[0], Nproto, Nch, Nslice);
    float* input = new float [Nch*(Nslice)];
    // hipfftComplex* inp_c = new hipfftComplex [Nch * Nslice / 2];
    hipfftComplex* output_gpu;
    hipfftComplex* output_cpu;
    output_cpu = new hipfftComplex [Nch*Nslice];
    hipMalloc((void **)&output_gpu, sizeof(hipfftComplex) * Nch * Nslice);
    hipHostRegister(input, sizeof(float)*Nch*Nslice, hipHostRegisterMapped);
    for (int k=0; k<2*Nsamples; k++)
    {
        float inp_arg = static_cast<float>(k / 2);
        if (k%2 == 0)
        {
            input[k] = sin(inp_arg);
        }
        else 
        {
            input[k] = sinc(2.0*inp_arg);
        }
    }
    cout << "---------------------------------------" << endl;
    float time;
    time_test(p_chann, input, output_gpu, 50, time);
    cout << "Channelization of " << Nsamples << " into 1024 channels takes " << time / 50 << " in milliseconds" << endl; 
    chann_destroy(p_chann);
    delete [] input;
    delete [] output_cpu;
    hipFree(output_gpu);

    // int Nproto = 128;
    // float kbeta = 10.2;
    // int Nch = 1024;
    // auto now = steady_clock::now();
    // for (int i=0; i<100; i++)
    // {
    //     float arg = Nproto / 2 + static_cast<float>(i + 1) / Nch;
    //     float darg = static_cast<float>(2 * i) / static_cast<float>(Nch*Nproto) - 1.0;
    //     float carg = kbeta * sqrt(1-darg*darg);
    //     float earg = cyl_bessel_if(0.0, carg) / cyl_bessel_if(0.0, kbeta);
    // }
    // auto end = steady_clock::now();
    // auto elapsed_time = duration<float, milli>(end - now).count();

    // cout << elapsed_time / 100 << endl;

    // auto box_collection = new box [16];

    // box* box_gpu;
    // hipMalloc((void**)&box_gpu, sizeof(box)*16);

    // hipMemcpy(box_gpu, box_collection, sizeof(box)*16, hipMemcpyHostToDevice);

    // test<<<4, 4>>>(box_gpu);
    // hipMemcpy(box_collection, box_gpu, sizeof(box)*16, hipMemcpyDeviceToHost);

    // for (int i=0; i< 16; i++)
    // {
    //     cout << box_collection[i].start_time << " " << box_collection[i].start_chann << " " << box_collection[i].stop_time << " " << box_collection[i].stop_chann << endl;
    // }

    // hipfftHandle* plans;
    // plans = new hipfftHandle [100];
    // auto istrides = new int [100];
    // auto ostrides = new int [100];
    // auto idists   = new int [100];
    // auto odists   = new int [100];
    // auto batches  = new int [100];
    // auto n = new int [100];
    
    // for (int i=0; i < 100; i++)
    // {
    //     istrides[i] = 2*i;
    //     ostrides[i] = 2*i;
    //     idists[i] = 4*i;
    //     odists[i] = 4*i;
    //     batches[i] = 1;
    //     n[i] = 2*(i%10+1);
    // }
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // float my_duration;
    // hipEventRecord(start);
    // for (int i=0; i<100; i++)
    // {
    //     hipfftPlanMany(&plans[i], 1, n + i, n+i, istrides[i], idists[i], n+i, ostrides[i], odists[i], HIPFFT_C2C, batches[i]);
    // }

    // for (int i=0; i<100; i++)
    // {
    //     hipfftDestroy(plans[i]);
    // }
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&my_duration, start, stop);

    // cout << "Initializing 100 plans and destroying them takes " << my_duration / 100 << " in milliseconds" << endl;
    
    // delete [] plans;
    // delete [] istrides;
    // delete [] ostrides;
    // delete [] idists;
    // delete [] odists;
    // delete [] batches;
    // delete [] n;

    // float* input;
    // hipMalloc((void**)&input, sizeof(float)*20);

    // float* output;
    // hipMalloc((void**)&output, sizeof(float)*20);

    // float* input_cpu = new float [20];
    // float* output_cpu = new float [20];

    // for (int i=0; i<20; i++)
    // {
    //     input_cpu[i] = static_cast<float>(i*i);
    // }

    // hipMemcpy(input, input_cpu, sizeof(float)*20, hipMemcpyHostToDevice);

    // hipMemcpy2D(output, 4, input + 5, 4, 2*sizeof(float), 5, hipMemcpyDeviceToDevice);
    // hipMemcpy(output_cpu, output, sizeof(float)*20, hipMemcpyDeviceToHost);

    // for (int i=0; i<20; i++)
    // {
    //     cout << output_cpu[i] << endl;
    // }

    // delete [] input_cpu;
    // hipFree(input);
    // hipFree(output);
    // delete [] output_cpu;

    // synth* revert_obj = synth_create(1024, 128, 32);

    // box* boxes = new box [100];

    // int nchannel = 1024;
    // int nslice   = 262144;

    // for (int i=0; i < 100; i++)
    // {
    //     boxes[i] = box(i, i+nslice / 32, 0, 512, i);
    // }


    // delete [] boxes;
    // synth_destroy(revert_obj);

}